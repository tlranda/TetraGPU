#include "hip/hip_runtime.h"
#include "cuda_extraction.h"

/* There are a few classes of functions here and general design patterns to be
   aware of:
    * "make_X_for_GPU" functions allocate GPU-side memory for X via a reference
        to a pointer on the caller's frame and use the host to populate
        appropriate values. No kernels, just remapping and data transfer.
    * "make_X_GPU" functions use GPU-side memory and kernels to create X using
        the GPU. Currently, these functions remap the memory to host and clean
        up the device allocation before exiting, but real applications may want
        the GPU-side relationship to persist (or to not be remapped in the same
        manner).
    * "X_kernel" functions are the device-side kernels that create new
        relationships using the GPU. They are called by the corresponding X's
        "make_X_GPU" kernel. All of the kernels attempt to utilize the GPU's
        thread scalability to get massive parallelism even if the linear
        algebra approach is very sparse. Our hope is that you can tie these
        functions together (perhaps as inlines) to make a device-contained
        relationship remap fit within the algorithm kernel with great locality.

  As a general note, everything is currently scheduled on the default stream;
  you may want to take care to adjust hipMemcpy and kernel invokations if
  overlapping is intended down the line.

  So far, we've found the "vectorized" kernel approach to be promising enough
  for relationship precompute. However, the approach is not trivial and
  requires several key strategies which you may want to re-use as you expand
  the relationship coverage.
    * "EV_kernel" shows a basic transpose from VE (split via "make_VE_for_GPU")
        and has coalesced reads with poor write ordering. You'll get one or the
        other to coalesce, and hardware generally optimizes for reads better.
    * "TF_kernel" shows a register-shuffle for exchanging global reads without
        shared memory. This ONLY works if you can do exchanges on powers-of-two
        exactly and get all necessary information. It then has a diverging
        block to handle thread-face assignment and a less-diverging scan lookup
        through the precomputed VF data to locate the correct face ID.
    * "TE_kernel" shows a shared-memory approach for exchanging global reads
        when it's necessary to do so on non-powers-of-two subsets. It also
        unrolls its relationship extraction to make more usage out of the
        requested shared memory. This unrolling pattern can technically be
        looped, but you have to adjust all constants to properly account for
        doing that (there are a bunch!). Pay special attention to definitions
        and comments that detail how these constants should be determined and
        adjusted for various constraints on the register file and occupancy.

  I've also run into a small number of strange behaviors here that are handled
  by workarounds for now; if you find a better way to do it, be vigilant to
  ensure all workarounds are replaced with the better behavior:
    * std::fill for host-side memory in "make_X_for_GPU" should work, but seems
        to segfault on certain sizes that our tests definitely reach and exceed.
        It may have something to do with working on pointers from
        CUDA_MALLOC_HOST, but root-causes were not identified and instead we
        see if the compiler understands the intent and can pitch in
        optimizations on the trivial loop with constant value assignments.
        Feel free to add #pragma unroll etc but I don't think they're present
        yet.
*/

void make_TV_for_GPU(vtkIdType ** device_tv,
                           // vector of array of vertices in a tetra
                     const TV_Data & tv_relationship) {
    // Size determination
    size_t tv_flat_size = sizeof(vtkIdType) * tv_relationship.nCells * nbVertsInCell;
    // Allocations
    CUDA_ASSERT(hipMalloc((void**)device_tv, tv_flat_size));
    vtkIdType * host_flat_tv = nullptr;
    CUDA_ASSERT(hipHostMalloc((void**)&host_flat_tv, tv_flat_size));

    // Set contiguous data in host memory
    vtkIdType index = 0;
    for (const auto & VertList : tv_relationship)
        for (const vtkIdType vertex : VertList)
            host_flat_tv[index++] = vertex;
    // Device copy and host free
    // BLOCKING -- provide barrier if made asynchronous to avoid free of host
    // memory before copy completes
    CUDA_WARN(hipMemcpy(*device_tv, host_flat_tv,
                         tv_flat_size, hipMemcpyHostToDevice));
    CUDA_WARN(hipHostFree(host_flat_tv));
}

void make_VE_for_GPU(vtkIdType ** device_vertices,
                     vtkIdType ** device_edges,
                     vtkIdType ** device_first_vertex,
                           // vector of vectors of edge IDs
                     const VE_Data & ve_relationship,
                     const vtkIdType n_verts,
                     const vtkIdType n_edges
                     ) {
    // Size determinations
    size_t vertices_size = sizeof(vtkIdType) * n_edges * nbVertsInEdge,
           // Can technically be half-sized, but duplicate for now so index in
           // vertices directly maps to edgeID without further manip (revisit
           // later as minor optimization -- would just drop multiplier of
           // nbVertsInEdge and adjust EVERY kernel making use of the edges
           // array to left-shift its index one bit (divide by two))
           edges_size = sizeof(vtkIdType) * n_edges * nbVertsInEdge,
           index_vertex_size = sizeof(vtkIdType) * n_verts;
    // Allocations
    CUDA_ASSERT(hipMalloc((void**)device_vertices, vertices_size));
    CUDA_ASSERT(hipMalloc((void**)device_edges, edges_size));
    CUDA_ASSERT(hipMalloc((void**)device_first_vertex, index_vertex_size));
    vtkIdType * host_vertices = nullptr,
              * host_edges = nullptr,
              * host_first_vertices = nullptr;
    CUDA_ASSERT(hipHostMalloc((void**)&host_vertices, vertices_size));
    CUDA_ASSERT(hipHostMalloc((void**)&host_edges, edges_size));
    CUDA_ASSERT(hipHostMalloc((void**)&host_first_vertices, index_vertex_size));

    Timer ve_translation;
    // Set contiguous data in host memory
    // Index defaults to END-OF-LIST to help with scanning
    // while std::fill should work, it can segfault on sizes (see the similar
    // code in make_VF_for_GPU() for further explanation; same bugfix should
    // apply here if one is found
    {
        const vtkIdType val = (n_verts+1)*nbVertsInEdge;
        for (vtkIdType i = 0; i < n_verts; i++) {
            host_first_vertices[i] = val;
        }
    }
    for (vtkIdType vertex_id = 0, index = 0, first = 0; vertex_id < n_verts; vertex_id++) {
        host_first_vertices[first++] = index;
        for (const EdgeData & edge : ve_relationship[vertex_id]) {
            // Pack low edge / ID
            host_edges[index] = edge.id;
            host_vertices[index++] = vertex_id;
            // Pack high edge / ID
            host_edges[index] = edge.id;
            host_vertices[index++] = edge.highVert;
        }
    }
    ve_translation.tick();

    // Device copy and host free
    // BLOCKING -- provide barrier if made asynchronous to avoid free of host
    // memory before copy completes
    ve_translation.tick();
    CUDA_WARN(hipMemcpy(*device_vertices, host_vertices,
                         vertices_size, hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(*device_edges, host_edges,
                         edges_size, hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(*device_first_vertex, host_first_vertices,
                         index_vertex_size, hipMemcpyHostToDevice));
    ve_translation.tick();
    ve_translation.label_interval(0, "VE Host->GPU Translation");
    ve_translation.label_interval(1, "VE Host->GPU Data Transfer");
    CUDA_WARN(hipHostFree(host_vertices));
    CUDA_WARN(hipHostFree(host_edges));
    CUDA_WARN(hipHostFree(host_first_vertices));
}

void make_VF_for_GPU(vtkIdType ** device_vertices,
                     vtkIdType ** device_faces,
                     vtkIdType ** device_first_faces,
                     const VF_Data & vf_relationship,
                     const vtkIdType n_verts,
                     const vtkIdType n_faces) {
    // Size determinations
    size_t vertices_size = sizeof(vtkIdType) * n_faces * nbVertsInFace,
           // Can technically be one-third this size, but duplicate for now
           faces_size =    sizeof(vtkIdType) * n_faces * nbVertsInFace,
           // Index into other arrays
           index_face_size = sizeof(vtkIdType) * n_verts;
    // Allocations
    CUDA_ASSERT(hipMalloc((void**)device_vertices, vertices_size));
    CUDA_ASSERT(hipMalloc((void**)device_faces, faces_size));
    CUDA_ASSERT(hipMalloc((void**)device_first_faces, index_face_size));
    vtkIdType * host_vertices = nullptr,
              * host_faces = nullptr,
              * host_first_faces = nullptr;
    CUDA_ASSERT(hipHostMalloc((void**)&host_vertices, vertices_size));
    CUDA_ASSERT(hipHostMalloc((void**)&host_faces, faces_size));
    CUDA_ASSERT(hipHostMalloc((void**)&host_first_faces, index_face_size));

    Timer vf_translation;
    // Set contiguous data in host memory
    // max_real_value = n_faces * nbVertsInFace
    // While std::fill should work, it can segfault on sizes that otherwise work?
    // KNOWN ISSUE: This size only appears to support up to 262,144 bytes allocation
    // in subsequent CUDA_MALLOC_HOST when the value is set to 800,000 bytes (100k vertices)
    // Not sure why -- may readdress later
    //std::fill(host_first_faces, host_first_faces+index_face_size, (n_faces+1) * nbVertsInFace);
    {
        const vtkIdType val = (n_faces+1)*nbVertsInFace;
        for (vtkIdType i = 0; i < n_verts; i++) {
            host_first_faces[i] = val;
        }
    }
    for (vtkIdType vertex_id = 0, index = 0; vertex_id < n_verts; vertex_id++) {
        for (const FaceData & face : vf_relationship[vertex_id]) {
            // Update first-face index if necessary
            if (host_first_faces[vertex_id] > index) host_first_faces[vertex_id] = index;
            // Pack lowest face / ID
            host_faces[index] = face.id;
            host_vertices[index++] = vertex_id;
            // Pack middle edge / ID
            host_faces[index] = face.id;
            host_vertices[index++] = face.middleVert;
            // Pack highest edge / ID
            host_faces[index] = face.id;
            host_vertices[index++] = face.highVert;
        }
    }
    // We use idx+1 to set a scan limit when looking for the face, though TBH
    // in C++/CUDA unless there's a bug in my logic we don't actually need to
    // set a scan limit as we'll find the face at before that value (and we can
    // always skip anything behind our first_face ID). But anyways, this will
    // ensure that our first_face values are monotonically increasing and if
    // we're able to do anything clever by knowing the scanning range, then you
    // merely look at the next element in the array to know when to stop.
    // We have to reverse-iterate the array to ensure sequences of 0-length
    // vertices are handled correctly, which is highly likely to occur a bunch
    // at the high-end of this data structure
    for (vtkIdType vertex_id = n_verts-1; vertex_id >= 0; vertex_id--) {
        if (host_first_faces[vertex_id] == (n_faces+1) * nbVertsInFace) {
            if (vertex_id == n_verts-1)
                host_first_faces[vertex_id] = n_faces * nbVertsInFace;
            else
                host_first_faces[vertex_id] = host_first_faces[vertex_id+1];
        }
    }
    vf_translation.tick();

    // Device copy and host free
    // BLOCKING -- provide barrier if made asynchronous to avoid free of host
    // memory before the copy completes
    vf_translation.tick();
    CUDA_WARN(hipMemcpy(*device_vertices, host_vertices, vertices_size,
                         hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(*device_faces, host_faces, faces_size,
                         hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(*device_first_faces, host_first_faces, index_face_size,
                         hipMemcpyHostToDevice));
    vf_translation.tick();
    vf_translation.label_interval(0, "VF Host->GPU Translation");
    vf_translation.label_interval(1, "VF Host->GPU Data Transfer");
    CUDA_WARN(hipHostFree(host_vertices));
    CUDA_WARN(hipHostFree(host_faces));
    CUDA_WARN(hipHostFree(host_first_faces));
}

__global__ void EV_kernel(const vtkIdType * __restrict__ vertices,
                          const vtkIdType * __restrict__ edges,
                          const vtkIdType n_edges,
                          vtkIdType * __restrict__ ev) {
    vtkIdType tid = (blockDim.x * blockIdx.x) + threadIdx.x,
              hi_vert = (tid % 2);
    if (tid >= (n_edges * nbVertsInEdge)) return;
    ev[(edges[tid] * nbVertsInEdge) + hi_vert] = vertices[tid];
}

// vector of array of vertices in an edge
                                     // vector of vector of EdgeData
std::unique_ptr<EV_Data> make_EV_GPU(const VE_Data & edgeTable,
                                     const vtkIdType n_points,
                                     const vtkIdType n_edges,
                                     const arguments args) {
    std::unique_ptr<EV_Data> edgeList = std::make_unique<EV_Data>();
    edgeList->reserve(n_edges);

    // Marshall data to GPU
    vtkIdType * vertices_device = nullptr,
              * edges_device = nullptr,
              * index_device = nullptr;
    make_VE_for_GPU(&vertices_device,
                    &edges_device,
                    &index_device,
                    edgeTable,
                    n_points,
                    n_edges
                    );
    // Free index_device as EV does not need it
    if (index_device != nullptr) CUDA_WARN(hipFree(index_device));
    // Compute the relationship
    size_t ev_size = sizeof(vtkIdType) * n_edges * nbVertsInEdge;
    vtkIdType * ev_computed = nullptr,
              * ev_host = nullptr;
    CUDA_ASSERT(hipMalloc((void**)&ev_computed, ev_size));
    CUDA_ASSERT(hipHostMalloc((void**)&ev_host, ev_size));
    vtkIdType n_to_compute = n_edges * nbVertsInEdge;
    dim3 thread_block_size = 1024,
         grid_size = (n_to_compute + thread_block_size.x - 1) / thread_block_size.x;
    std::cout << INFO_EMOJI << "Kernel launch configuration is " << grid_size.x
              << " grid blocks with " << thread_block_size.x
              << " threads per block" << std::endl;
    std::cout << INFO_EMOJI << "The mesh has " << n_points << " points and "
              << n_edges << " edges" << std::endl;
    std::cout << INFO_EMOJI << "Tids >= " << n_edges * nbVertsInEdge
              << " should auto-exit ("
              << (thread_block_size.x * grid_size.x) - n_to_compute << ")"
              << std::endl;
    Timer kernel;
    KERNEL_WARN(EV_kernel<<<grid_size KERNEL_LAUNCH_SEPARATOR
                            thread_block_size>>>(vertices_device,
                                edges_device,
                                n_edges,
                                ev_computed));
    CUDA_WARN(hipDeviceSynchronize());
    kernel.tick();
    kernel.label_prev_interval("GPU kernel duration");
    // Copy back to host and set in edgeList
    kernel.tick();
    CUDA_WARN(hipMemcpy(ev_host, ev_computed, ev_size, hipMemcpyDeviceToHost));
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host transfer");
    kernel.tick();
    // Reconfigure into edgeList for comparison
    #pragma omp parallel for num_threads(args.threadNumber)
    for (vtkIdType e = 0; e < n_edges; ++e)
        edgeList->emplace_back(std::array<vtkIdType,nbVertsInEdge>{
                                    ev_host[(2*e)],ev_host[(2*e)+1]});
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host translation");
    // Free device memory
    if (vertices_device != nullptr) CUDA_WARN(hipFree(vertices_device));
    if (edges_device != nullptr) CUDA_WARN(hipFree(edges_device));
    if (ev_computed != nullptr) CUDA_WARN(hipFree(ev_computed));
    // Free host memory
    if (ev_host != nullptr) CUDA_WARN(hipHostFree(ev_host));

    return edgeList;
}

__global__ void TF_kernel(const vtkIdType * __restrict__ tv,
                          const vtkIdType * __restrict__ vertices,
                          const vtkIdType * __restrict__ faces,
                          const vtkIdType * __restrict__ first_faces,
                          const vtkIdType n_cells,
                          const vtkIdType n_faces,
                          const vtkIdType n_points,
                          vtkIdType * __restrict__ tf) {
    vtkIdType tid = (blockDim.x * blockIdx.x) + threadIdx.x,
              face = (tid % 4);
    if (tid >= (n_cells * nbFacesInCell)) return;

    // Read your TV value -- because there are 4 vertices in a cell, every warp
    // is automatically cell-aligned in memory along 8 cells :)
    vtkIdType cell_vertex = tv[tid], v0, v1, v2, v3;
    // Use register exchanges within the warp to read all other values for your
    // cell
    v0 = __shfl_sync(0xffffffff, cell_vertex, 0, 4);
    v1 = __shfl_sync(0xffffffff, cell_vertex, 1, 4);
    v2 = __shfl_sync(0xffffffff, cell_vertex, 2, 4);
    v3 = __shfl_sync(0xffffffff, cell_vertex, 3, 4);

    // !! Define each TID's represented face -- divergence expected !!
    /*
       f0 = v0 - v1 - v2
       f1 = v1 - v2 - v3
       f2 = v0 - v2 - v3
       f3 = v0 - v1 - v3
    */
    vtkIdType face_low = v0, face_mid = v1, face_high = v3;
    if (face == 0) face_high = v2;
    if (face == 1 || face == 2) {
        face_mid = v2;
        if (face == 1) face_low = v1;
    }

    // While syncing may not be strictly required, empirically it seems to make
    // the kernel faster on average
    __syncthreads();

    // !! Scan VF for your face match -- divergence expected !!
    // We do NOT guard against an out-of-bounds check on the condition, as the
    // LOW face explicitly has 2 other vertices higher than it, therefore those
    // vertices always define an upper bound without touching OOM (worst-case
    // those vertices indicate to go to the end of the vertices array)
    for (vtkIdType i = first_faces[face_low]; i < first_faces[face_low+1]; i += 3) {
        if (vertices[i+1] == face_mid && vertices[i+2] == face_high) {
            tf[tid] = faces[i];
            break;
        }
        __syncthreads();
    }
}

std::unique_ptr<TF_Data> make_TF_GPU(const TV_Data & TV,
                                     const VF_Data & VF,
                                     const vtkIdType n_points,
                                     const vtkIdType n_faces,
                                     const vtkIdType n_cells,
                                     const arguments args) {
    std::unique_ptr<TF_Data> TF = std::make_unique<TF_Data>();
    TF->reserve(n_cells);

    // Make data ready for GPU
    vtkIdType * tv_device = nullptr,
              * vertices_device = nullptr,
              * faces_device = nullptr,
              * first_faces_device = nullptr;
    make_TV_for_GPU(&tv_device, TV);
    make_VF_for_GPU(&vertices_device, &faces_device, &first_faces_device, VF,
                    n_points, n_faces);

    // Compute the relationship
    size_t tf_size = sizeof(vtkIdType) * n_cells * nbFacesInCell;
    vtkIdType * tf_computed = nullptr,
              * tf_host = nullptr;
    CUDA_ASSERT(hipMalloc((void**)&tf_computed, tf_size));
    CUDA_ASSERT(hipHostMalloc((void**)&tf_host, tf_size));
    vtkIdType n_to_compute = n_cells * nbFacesInCell;
    dim3 thread_block_size = 1024,
         grid_size = (n_to_compute + thread_block_size.x - 1) / thread_block_size.x;
    std::cout << INFO_EMOJI << "Kernel launch configuration is " << grid_size.x
              << " grid blocks with " << thread_block_size.x << " threads per block"
              << std::endl;
    std::cout << INFO_EMOJI << "The mesh has " << n_cells << " cells and "
              << n_faces << " faces" << std::endl;
    std::cout << INFO_EMOJI << "Tids >= " << n_cells * nbFacesInCell << " should auto-exit ("
              << (thread_block_size.x * grid_size .x) - n_to_compute << ")"
              << std::endl;
    Timer kernel;
    KERNEL_WARN(TF_kernel<<<grid_size KERNEL_LAUNCH_SEPARATOR
                            thread_block_size>>>(tv_device,
                                vertices_device,
                                faces_device,
                                first_faces_device,
                                n_cells,
                                n_faces,
                                n_points,
                                tf_computed));
    CUDA_WARN(hipDeviceSynchronize());
    kernel.tick();
    kernel.label_prev_interval("GPU kernel duration");
    // Copy back to host and set in edgeList
    kernel.tick();
    CUDA_WARN(hipMemcpy(tf_host, tf_computed, tf_size, hipMemcpyDeviceToHost));
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host transfer");
    kernel.tick();
    // Reconfigure into host-side structure for comparison
    for (vtkIdType c = 0; c < n_cells; ++c) {
        TF->emplace_back(std::array<vtkIdType,nbFacesInCell>{
                tf_host[(nbFacesInCell*c)],
                tf_host[(nbFacesInCell*c)+1],
                tf_host[(nbFacesInCell*c)+2],
                tf_host[(nbFacesInCell*c)+3],
                });
    }
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host translation");
    // Free device memory
    if (tv_device != nullptr) CUDA_WARN(hipFree(tv_device));
    if (vertices_device != nullptr) CUDA_WARN(hipFree(vertices_device));
    if (faces_device != nullptr) CUDA_WARN(hipFree(faces_device));
    if (first_faces_device != nullptr) CUDA_WARN(hipFree(first_faces_device));
    if (tf_computed != nullptr) CUDA_WARN(hipFree(tf_computed));
    if (tf_host != nullptr) CUDA_WARN(hipHostFree(tf_host));
    return TF;
}

__device__ __inline__ void te_combine(vtkIdType quad0, vtkIdType quad1,
                                      vtkIdType quad2, vtkIdType quad3,
                                      const vtkIdType laneID,
                                      vtkIdType * __restrict__ te,
                                      const vtkIdType * __restrict__ vertices,
                                      const vtkIdType * __restrict__ edges,
                                      const vtkIdType n_points,
                                      const vtkIdType * __restrict__ index) {
    // Within each sub-group, assign unique combination of vertex pairs from quad
    // Then look up the edge ID in VE and assign it to TE
    /* Pattern:
       0: q0 - q1
       1: q1 - q2
       2: q2 - q3
       3: q0 - q2
       4: q1 - q3
       5: q0 - q3
    */
    vtkIdType left_vertex = quad0, // 0, 3, 5
              right_vertex = quad3; // 2, 4, 5
    if (laneID == 1 || laneID == 2 || laneID == 4) {
        if (laneID == 2) left_vertex = quad2;
        else /* 1, 4 */ left_vertex = quad1;
    }
    if (laneID == 0 || laneID == 1 || laneID == 3) {
        if (laneID == 0) right_vertex = quad1;
        else /* 1, 3 */ right_vertex = quad2;
    }
    // Ensure lowest index is the left one
    if (left_vertex > right_vertex) {
        vtkIdType swap = left_vertex;
        left_vertex = right_vertex;
        right_vertex = swap;
    }

    __syncthreads();
    // !! Scan VE for first edge match -- divergence expected !!
    // There is no OOB guard on the for-loop condition as the LOWER index is
    // explicitly less than the HIGHER index, ergo index[left_vertex+1] is
    // definitely in-bounds
    for (vtkIdType i = index[left_vertex]; i < index[left_vertex+1]; i+= 2) {
        // vertices = [low-edge, high-edge] x n-Edges
        // edges =    [edge id , edge id  ] x n-Edges
        if (vertices[i+1] == right_vertex) {
            // TE is already shifted for every thread, so just write to your
            // laneID and that should mark the edge
            te[laneID] = edges[i];
            break;
        }
    }
    __syncthreads();
}

#define TE_CELLS_PER_BLOCK 195
__global__ void TE_kernel(const vtkIdType * __restrict__ tv,
                          const vtkIdType * __restrict__ vertices,
                          const vtkIdType * __restrict__ edges,
                          const vtkIdType * __restrict__ first_index,
                          const vtkIdType n_cells,
                          const vtkIdType n_edges,
                          const vtkIdType n_points,
                          vtkIdType * __restrict__ te) {
    // LAUNCH WITH 6 THREADS PER CELL, LOSE 2 THREADS PER WARP (32) WHICH
    // REQUIRES OVERSUBSCRIPTION IMMEDIATELY
    // ALSO MUST ALLOCATE ENOUGH SHARED MEMORY FOR KERNEL.

    // TAKE CARE THAT CONSTANTS ARE WRITTEN FOR UNROLLING 3 LOOP ITERATIONS,
    // IF UNROLLING MORE OR LESS, THESE CONSTANTS MUST BE UPDATED
    extern __shared__ vtkIdType sh_scratch[];

    vtkIdType tid = (blockIdx.x * blockDim.x) + threadIdx.x,
              warpID = (threadIdx.x % 32),
              laneID = warpID % 6,
              laneDepth = 3*(((tid / 32)*5) + (warpID / 6)),
              /* shLaneDepth MUST ALWAYS BE MODULO THE CELLS_PER_BLOCK VALUE */
              shLaneDepth = laneDepth % TE_CELLS_PER_BLOCK,
              edge = (tid % nbEdgesInCell);
    // Early-exit threads reading beyond #cells at base index AND 2 straggler threads per warp
    if (laneDepth >= n_cells || warpID > 29) return;

    // Push output pointer TE per-thread to its writing position
    te += (laneDepth * 6);

    // Read FIRST value from global memory --> shared
    // laneDepth *= 4 to use vector-addressing; not set permanently as the
    // cellID is nice to hold onto for later
    vtkIdType read_indicator = n_cells-laneDepth-1;
    if (read_indicator >= 1 || (read_indicator == 0 && laneID < 4)) {
        sh_scratch[(shLaneDepth*6)+laneID] = tv[(laneDepth*4)+laneID];
    }
    __syncthreads();

    // UNROLL 1: First quadruplet is guaranteed to be useful due to early-exit threads no longer being present
    vtkIdType quad0 = sh_scratch[(shLaneDepth*6)  ],
              quad1 = sh_scratch[(shLaneDepth*6)+1],
              quad2 = sh_scratch[(shLaneDepth*6)+2],
              quad3 = sh_scratch[(shLaneDepth*6)+3];
    // All 6 combinations of values need to be made to get the TE relationship,
    // but the edgeID has to be looked up in VE relationship
    te_combine(quad0,quad1,quad2,quad3, laneID, te, vertices, edges, n_points,
               first_index);

    // UNROLL 2: Second quadruplet is half-read already; exit if NOT useful
    if (read_indicator == 0) return;
    // Adjust pointers to not overwrite previous iteration's data
    te += 6;
    quad0 = sh_scratch[(shLaneDepth*6)+4];
    quad1 = sh_scratch[(shLaneDepth*6)+5];
    __syncthreads();
    // Continue reading for unrolls 2 & 3
    if (read_indicator > 1 || (read_indicator == 1 & laneID < 2)) {
        sh_scratch[(shLaneDepth*6)+laneID] = tv[(laneDepth*4)+laneID+6];
    }
    __syncthreads();
    quad2 = sh_scratch[(shLaneDepth*6)  ];
    quad3 = sh_scratch[(shLaneDepth*6)+1];
    te_combine(quad0,quad1,quad2,quad3, laneID, te, vertices, edges, n_points,
               first_index);

    // UNROLL 3: Third quadruplet is read; early exit if NOT useful
    if (read_indicator == 1) return;
    // Adjust pointers to not overwrite previous iteration's data
    te += 6;
    quad0 = sh_scratch[(shLaneDepth*6)+2];
    quad1 = sh_scratch[(shLaneDepth*6)+3];
    quad2 = sh_scratch[(shLaneDepth*6)+4];
    quad3 = sh_scratch[(shLaneDepth*6)+5];
    te_combine(quad0,quad1,quad2,quad3, laneID, te, vertices, edges, n_points,
               first_index);
}
// TE = TV x VE
std::unique_ptr<TE_Data> make_TE_GPU(const TV_Data & TV,
                                     const VE_Data & VE,
                                     const vtkIdType n_points,
                                     const vtkIdType n_edges,
                                     const vtkIdType n_cells,
                                     const arguments args) {
    std::unique_ptr<TE_Data> TE = std::make_unique<TE_Data>();
    TE->reserve(n_cells);

    // Make ready for GPU
    vtkIdType * tv_device = nullptr,
              * vertices_device = nullptr,
              * edges_device = nullptr,
              * index_device = nullptr;
    make_TV_for_GPU(&tv_device, TV);
    make_VE_for_GPU(&vertices_device,
                    &edges_device,
                    &index_device,
                    VE,
                    n_points,
                    n_edges
                    );

    // Compute relationship
    vtkIdType n_to_compute = n_cells * nbEdgesInCell;
    size_t te_size = sizeof(vtkIdType) * n_to_compute;
    vtkIdType * te_computed = nullptr,
              * te_host = nullptr;
    CUDA_ASSERT(hipMalloc((void**)&te_computed, te_size));
    CUDA_ASSERT(hipHostMalloc((void**)&te_host, te_size));

    // Set up launch configuration for the kernel
    const vtkIdType N_THREADS = 416,
                    /*
                       6 edges required per cell (1 edge : 1 thread)
                       Up to 3 cells unrolled in each group of threads
                       -2 threads per warp of 32 threads for warp alignment on factor of 6
                       480 cells comes from:
                       6*((480+2)//3) == 6 * 160 = 960 work with unrolling
                       ((960+29)//30)*32 == 32*32 = 1024 threads in block

                       Max 1024 threads per block in hardware, increasing to 481 threads requires a new block for the warp

                       -- however, register usage can pose an even greater problem for us --

                       1024 threads * 78 registers (current HW) = 79,872 / 65,536 registers demanded
                       Our early-exits cost us in that the CUDA launch API has
                       no clue that we're going to honor that

                       At 78 registers, we can use up to 840 threads in a block
                       Round this down to 832 == 26*32 (fullwarp alignment)
                       Each warp has 5 groups (26*5 == 130 single-cells), with
                       3 unrolled for 390 cells per block after unrolling

                       The above isn't working on this hardware, idk let's cut
                       it in half. 416 threads -> 13 full warps AKA 65 groups
                       unrolling to 195 cells in a block

                       The value ALSO needs to be set within the kernel, so
                       update the TE_CELLS_PER_BLOCK preprocessor definition
                       above the TE_kernel() function if you need to change it
                    */
                    cells_per_block = TE_CELLS_PER_BLOCK,
                    SHARED_PER_BLOCK = cells_per_block * 6 * sizeof(vtkIdType);
    vtkIdType N_BLOCKS = (n_cells+cells_per_block-1)/cells_per_block;

    std::cout << INFO_EMOJI << "Kernel launch configuration is " << N_BLOCKS
              << " grid blocks with " << N_THREADS << " threads per block"
              << " and " << SHARED_PER_BLOCK << " bytes shmem per block"
              << std::endl;
    std::cout << INFO_EMOJI << "The mesh has " << n_cells << " cells and "
              << n_edges << " edges" << std::endl;
    if (hipFuncSetAttribute(reinterpret_cast<const void*>(TE_kernel),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             49152/*SHARED_PER_BLOCK*/) != hipSuccess) {
        std::cerr << WARN_EMOJI << "Could not set max dynamic shared memory size to "
                  << SHARED_PER_BLOCK << " bytes" << std::endl;
    }
    Timer kernel;
    KERNEL_WARN(TE_kernel<<<N_BLOCKS KERNEL_LAUNCH_SEPARATOR
                            N_THREADS KERNEL_LAUNCH_SEPARATOR
                            SHARED_PER_BLOCK>>>(tv_device,
                                vertices_device,
                                edges_device,
                                index_device,
                                n_cells,
                                n_edges,
                                n_points,
                                te_computed));
    CUDA_WARN(hipDeviceSynchronize());
    kernel.tick();
    kernel.label_prev_interval("GPU kernel duration");

    // Copy back to host with transformation
    kernel.tick();
    CUDA_WARN(hipMemcpy(te_host, te_computed, te_size, hipMemcpyDeviceToHost));
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host transfer");
    kernel.tick();
    // Reconfigure for host
    for (vtkIdType c = 0; c < n_cells; ++c) {
        TE->emplace_back(std::array<vtkIdType,nbEdgesInCell>{
                te_host[(6*c)  ], te_host[(6*c)+1],
                te_host[(6*c)+2], te_host[(6*c)+3],
                te_host[(6*c)+4], te_host[(6*c)+5]
                });
    }
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host translation");
    // Free device memory
    if (tv_device != nullptr) CUDA_WARN(hipFree(tv_device));
    if (vertices_device != nullptr) CUDA_WARN(hipFree(vertices_device));
    if (edges_device != nullptr) CUDA_WARN(hipFree(edges_device));
    if (te_computed != nullptr) CUDA_WARN(hipFree(te_computed));
    if (te_host != nullptr) CUDA_WARN(hipHostFree(te_host));
    return TE;
}

__global__ void FV_kernel(const vtkIdType * __restrict__ vertices,
                          const vtkIdType * __restrict__ faces,
                          const vtkIdType n_faces,
                          vtkIdType * __restrict__ fv) {
    vtkIdType tid = (blockDim.x * blockIdx.x) + threadIdx.x,
              vert_idx = (tid % 3);
    if (tid >= (n_faces * nbVertsInFace)) return;
    fv[(faces[tid] * nbVertsInFace) + vert_idx] = vertices[tid];
}

std::unique_ptr<FV_Data> make_FV_GPU(const VF_Data & VF,
                                     const vtkIdType n_points,
                                     const vtkIdType n_faces,
                                     const arguments args) {
    // FV_data = std::vector<FaceData{middleVert,highVert,id}>
    std::unique_ptr<FV_Data> vertexList = std::make_unique<FV_Data>();
    vertexList->reserve(n_faces);

    // Marshall data to GPU
    vtkIdType * vertices_device = nullptr,
              * faces_device = nullptr,
              * index_device = nullptr;
    make_VF_for_GPU(&vertices_device,
                    &faces_device,
                    &index_device,
                    VF,
                    n_points,
                    n_faces
                    );
    // Free index device as FV does not need it
    if (index_device != nullptr) CUDA_WARN(hipFree(index_device));
    // Compute the relationship
    size_t fv_size = sizeof(vtkIdType) * n_faces * nbVertsInFace;
    vtkIdType * fv_computed = nullptr,
              * fv_host = nullptr;
    CUDA_ASSERT(hipMalloc((void**)&fv_computed, fv_size));
    CUDA_ASSERT(hipHostMalloc((void**)&fv_host, fv_size));
    vtkIdType n_to_compute = n_faces * nbVertsInFace;
    dim3 thread_block_size = 1024,
         grid_size = (n_to_compute + thread_block_size.x - 1) / thread_block_size.x;
    std::cout << INFO_EMOJI << "Kernel launch configuration is " << grid_size.x
              << " grid blocks with " << thread_block_size.x
              << " threads per block" << std::endl;
    std::cout << INFO_EMOJI << "The mesh has " << n_points << " points and "
              << n_faces << " faces" << std::endl;
    std::cout << INFO_EMOJI << "Tids >= " << n_faces * nbVertsInFace
              << " should auto-exit ("
              << (thread_block_size.x * grid_size.x) - n_to_compute << ")"
              << std::endl;
    Timer kernel;
    KERNEL_WARN(FV_kernel<<<grid_size KERNEL_LAUNCH_SEPARATOR
                            thread_block_size>>>(vertices_device,
                                faces_device,
                                n_faces,
                                fv_computed));
    CUDA_WARN(hipDeviceSynchronize());
    kernel.tick();
    kernel.label_prev_interval("GPU kernel duration");
    // Copy back to host and set in vertexList
    kernel.tick();
    CUDA_WARN(hipMemcpy(fv_host, fv_computed, fv_size, hipMemcpyDeviceToHost));
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host transfer");
    kernel.tick();
    // Reconfigure for host comparison
    //#pragma omp parallel for num_threads(args.threadNumber)
    for (vtkIdType f = 0; f < n_faces; ++f) {
        vertexList->emplace_back(FaceData(fv_host[(3*f)], fv_host[(3*f)+1],
                                          fv_host[(3*f)+2]));
    }
    kernel.tick();
    kernel.label_prev_interval("GPU Device->Host translation");
    // Free device memory
    if (vertices_device != nullptr) CUDA_WARN(hipFree(vertices_device));
    if (faces_device != nullptr) CUDA_WARN(hipFree(faces_device));
    if (fv_computed != nullptr) CUDA_WARN(hipFree(fv_computed));
    // Free host memory
    if (fv_host != nullptr) CUDA_WARN(hipHostFree(fv_host));

    return vertexList;
}

