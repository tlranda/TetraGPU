#include "hip/hip_runtime.h"
#include "alg/critPoints.h"

/* Drives a critical points algorithm on an unstructured tetrahedral mesh. This
   can generalize to higher-order meshes if they are preprocessed to be divided
   into tetrahedra (not included).

   Mesh data is loaded via the VTK unstructured format (.vtu). Only scalar data
   is supported on the mesh at this time (not vectors on the mesh). We only
   classify points as regular, maximum, minimum, or saddles (we do not inspect
   sub-classes of saddles).
*/

// This kernel helps to ensure consistent and accurate timing of device-side
// events
__global__ void dummy_kernel(void) {
    //int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
}

/* CriticalPoints kernel should:
        1) Parallelize VV on second-dimension (can early-exit block if no data
           available or if a prefix-scan of your primary-dimension list shows
           that you are a duplicate)
        2) Read the scalar value used for point classification and classify
           yourself relative to your primary-dimension scalar value as upper or
           lower neighbor
        -- VV-PARALLEL SYNC REQUIRED --
        3) For all other threads sharing your neighborhood classification, scan
           their connectivity in VV. If you connect to at least one, then you
           share a component with that neighbor -- the lowest-ranked neighbor
           will log +1 component of this type and all others exit. If you fail
           to locate any connections to others in your class, then you have 2+
           components and are immediately a saddle -- increment your component
           counter and exit. It does not matter if this "over-counts" the
           number of components!
        -- VV-PARALLEL SYNC REQUIRED --
        4) Classification is performed as follows: Exactly 1 upper component is
           a maximum; exactly 1 lower component is a minimum; two or more upper
           or lower components is a saddle; other values are regular.
*/
__global__ void critPoints(const vtkIdType * __restrict__ VV,
                           const unsigned long long * __restrict__ VV_index,
                           vtkIdType * __restrict__ valences,
                           const vtkIdType points,
                           const vtkIdType max_VV_guess,
                           const double * __restrict__ scalar_values,
                           unsigned int * __restrict__ classes) {
    vtkIdType tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    /*
        1) Parallelize VV on second-dimension (can early-exit block if no data
           available or if a prefix-scan of your primary-dimension list shows
           that you are a duplicate)
    */
    // Beyond scope of work for the kernel
    if (tid >= (points * max_VV_guess)) return;
    vtkIdType my_1d = tid / max_VV_guess,
              my_2d = VV[tid];
    // No work for this point's valence
    if (VV_index[my_1d] <= 0) return;
    // Prefix scan as anti-duplication
    for (vtkIdType i = my_1d * max_VV_guess; i < tid; i++) {
        if (VV[i] == my_2d) return;
    }

    // BEYOND THIS POINT, YOU ARE AN ACTUAL WORKER THREAD ON THE PROBLEM

    /*
        2) Read the scalar value used for point classification and classify
           yourself relative to your primary-dimension scalar value as upper or
           lower neighbor
        -- VV-PARALLEL SYNC REQUIRED --
    */
    // Classify yourself as an upper or lower valence neighbor to your 1d point
    // Upper = -1, Lower = 1
    //vtkIdType my_class = 1 - ((scalar_values[my_2d] >= scalar_values[my_1d])<<1);
    vtkIdType my_class = 1 - ((scalar_values[my_2d] < scalar_values[my_1d])<<1);
    valences[tid] = my_class;
    __syncthreads();
    /*
        3) For all other threads sharing your neighborhood classification, scan
           their connectivity in VV. If you connect to at least one, then you
           share a component with that neighbor -- the lowest-ranked neighbor
           will log +1 component of this type and all others exit. If you fail
           to locate any connections to others in your class, then you have 2+
           components and are immediately a saddle -- increment your component
           counter and exit. It does not matter if this "over-counts" the
           number of components!
        -- VV-PARALLEL SYNC REQUIRED --
    */
    vtkIdType max_my_1d = (my_1d * max_VV_guess) + VV_index[my_1d];
    bool done = false;
    for(vtkIdType i = my_1d * max_VV_guess; !done && (i < max_my_1d); i++) {
        if (valences[i] == my_class) {
            // Find yourself in their adjacency to become a shared component
            vtkIdType max_my_2d = VV[i] + VV_index[i / max_VV_guess];
            for(vtkIdType j = VV[i]; !done && (j < max_my_2d); j++) {
                if (VV[j] == my_2d) {
                    // Shared component!
                    // Upper == -1, (-1+1)/2 => 0
                    // Lower ==  1, (1+1)/2  => 1
                    // --no atomic for vtkIdType-- atomicAdd(classes[(my_1d*3) + ((my_class+1)/2)],1);
                    // best match: unsigned long long int
                    // other matches: unsigned int, int
                    //classes[(my_1d*3) + ((my_class+1)/2)] += 1;
                    atomicAdd(classes+((my_1d*3) + ((my_class+1)/2)), 1);
                    // Break all loops
                    done = true;
                }
            }
        }
    }
    __syncthreads();
    /*
        4) Classification is performed as follows: Exactly 1 upper component is
           a maximum; exactly 1 lower component is a minimum; two or more upper
           or lower components is a saddle; other values are regular.
    */
    // Limit classification to lowest-ranked thread for single write
    if (my_1d * max_VV_guess == tid) {
        vtkIdType upper = classes[(my_1d*3)],
                  lower = classes[(my_1d*3)+1];
        if (upper == 1 && lower == 0) classes[(my_1d*3)+2] = 1; // Maximum
        else if (upper == 0 && lower == 1) classes[(my_1d*3)+2] = 2; // Minimum
        else if (upper == 1 && lower == 1) classes[(my_1d*3)+2] = 3; // Regular
        else classes[(my_1d*3)+2] = 4; // Saddle
    }
}

void export_classes(unsigned int * classes, vtkIdType n_classes, arguments & args) {
    std::ofstream output_fstream; // Used for file handle to indicated name
    std::streambuf * output_buffer; // Buffer may point to stdout or file handle
    if (args.export_ == "") {
        // No export provided by user, write to stdout
        output_buffer = std::cout.rdbuf();
        std::cerr << WARN_EMOJI << YELLOW_COLOR << "No export file; outputting "
                  "classes to stdout" << RESET_COLOR << std::endl;
    }
    else {
        // Put results in the indicated file
        output_fstream.open(args.export_);
        output_buffer = output_fstream.rdbuf();
        std::cerr << INFO_EMOJI << "Outputting classes to " << args.export_
                                << std::endl;
    }
    // Used for actual file handling
    std::ostream out(output_buffer);
    std::string class_names[] = {"NULL", "max", "min", "regular", "saddle"};
    vtkIdType n_insane = 0;
    for (vtkIdType i = 0; i < n_classes; i++) {
        // The classification information is provided, then the class:
        // {# upper, # lower, class}
        // CLASSES = {'maximum': 1, 'minimum': 2, 'regular': 3, 'saddle': 4}
        unsigned int my_class = classes[(i*3)+2],
                     n_upper  = classes[(i*3)],
                     n_lower  = classes[(i*3)+1];
        // Misclassification sanity checks
        if ((n_lower == 0 && n_upper == 1 && my_class != 1) ||
            (n_lower == 1 && n_upper == 0 && my_class != 2) ||
            (n_lower == 1 && n_upper == 1 && my_class != 3) ||
            (n_lower != 1 && n_upper != 1 && my_class != 4)) {
            out << "INSANITY DETECTED FOR POINT " << i << std::endl;
            n_insane++;
        }
        out << "Class " << i << " = " << my_class << std::endl;
        //out << "Class " << i << " = " << class_names[my_class] << std::endl;
    }
    if (n_insane > 0) {
        std::cerr << WARN_EMOJI << RED_COLOR << "Insanity detected; "
                     "GPU did not agree on its own answers for " << n_insane
                  << " points." << RESET_COLOR << std::endl;
    }
    #ifdef VALIDATE_GPU
    else {
        std::cerr << OK_EMOJI << "No insanity detected in GPU self-agreement "
                     "when classifying points." << std::endl;
    }
    #endif
}

int main(int argc, char *argv[]) {
    Timer timer(false, "Main");
    arguments args;
    parse(argc, argv, args);
    timer.tick();
    timer.interval("Argument parsing");

    // GPU initialization
    if (! args.validate()) {
        timer.label_next_interval("GPU context creation with dummy kernel");
        timer.tick();
        KERNEL_WARN(dummy_kernel<<<1 KERNEL_LAUNCH_SEPARATOR 1>>>());
        CUDA_ASSERT(hipDeviceSynchronize());
        timer.tick_announce();
        timer.label_next_interval("GPU trivial kernel launch");
        timer.tick();
        KERNEL_WARN(dummy_kernel<<<1 KERNEL_LAUNCH_SEPARATOR 1>>>());
        CUDA_ASSERT(hipDeviceSynchronize());
        timer.tick_announce();
    }

    // MANDATORY: TV (green) [from storage]
    std::cout << PUSHPIN_EMOJI << "Parsing vtu file: " << args.fileName
              << std::endl;
    timer.label_next_interval(GREEN_COLOR "TV" RESET_COLOR " from VTK");
    timer.tick();
    // Should utilize VTK API and then de-allocate all of its heap
    // Also loads the vertex attributes (host-side) and sets them in
    // TV->vertexAttributes (one scalar per vertex)
    std::unique_ptr<TV_Data> TV = get_TV_from_VTK(args); // args.filename
    timer.tick_announce();

    // Usually VE and VF are also mandatory, but CritPoints does not require
    // these relationships! Skip them!

    // OPTIONAL: VV (yellow) [TV' x TV]
    // REQUIRED for CritPoints
    std::cout << PUSHPIN_EMOJI << "Using GPU to compute " YELLOW_COLOR "VV" RESET_COLOR << std::endl;
    timer.label_next_interval(YELLOW_COLOR "VV" RESET_COLOR " [GPU]");
    timer.tick();
    // Have to make a max VV guess
    vtkIdType max_VV_guess = get_approx_max_VV(*TV, TV->nPoints);
    device_VV * dvv = make_VV_GPU_return(*TV, TV->nCells, TV->nPoints,
                                         max_VV_guess, true, args); // Args not used, actually
    timer.tick_announce();

    // Critical Points
    timer.label_next_interval("Allocate " CYAN_COLOR "Critical Points" RESET_COLOR " memory");
    timer.tick();
    // CPC = actual critical points classifications
    // valences = adjacency upper/lower classification PRIOR to point classification
    unsigned int *host_CPC = nullptr,
                 *device_CPC = nullptr;
    vtkIdType *device_valences = nullptr,
              *scalar_values = nullptr;
    double    *device_scalar_values = nullptr;
    // #Upper, #Lower, Classification
    size_t classes_size = sizeof(unsigned int) * TV->nPoints * 3,
           // Upper/lower per adjacency
           valences_size = sizeof(vtkIdType) * TV->nPoints * max_VV_guess,
           scalars_size = sizeof(double) * TV->nPoints;
    CUDA_ASSERT(hipHostMalloc((void**)&host_CPC, classes_size));
    CUDA_ASSERT(hipMalloc((void**)&device_CPC, classes_size));
    CUDA_ASSERT(hipMalloc((void**)&device_valences, valences_size));
    CUDA_ASSERT(hipHostMalloc((void**)&scalar_values, scalars_size));
    CUDA_ASSERT(hipMalloc((void**)&device_scalar_values, scalars_size));
    // Pre-populate valences as zeros and populate scalar values
    {
        vtkIdType * valences = nullptr;
        CUDA_ASSERT(hipHostMalloc((void**)&valences, valences_size));
        for(vtkIdType i = 0; i < valences_size / sizeof(vtkIdType); i++) {
            valences[i] = 0;
        }
        CUDA_WARN(hipMemcpy(device_valences, valences, valences_size, hipMemcpyHostToDevice));
        if (valences != nullptr) CUDA_WARN(hipHostFree(valences));

        // Scalar values from VTK
        for(vtkIdType i = 0; i < TV->nPoints; i++) {
            scalar_values[i] = TV->vertexAttributes[i];
        }
        CUDA_WARN(hipMemcpy(device_scalar_values, scalar_values, scalars_size, hipMemcpyHostToDevice));
    }
    timer.tick_announce();
    timer.label_next_interval("Run " CYAN_COLOR "Critical Points" RESET_COLOR " algorithm");
    // Set kernel launch parameters here
    /*
        1) Parallelize VV on second-dimension (can early-exit block if no data
           available or if a prefix-scan of your primary-dimension list shows
           that you are a duplicate)
    */
    const vtkIdType n_to_compute = TV->nPoints * max_VV_guess;
    dim3 thread_block_size = max_VV_guess,
         grid_size = (n_to_compute + thread_block_size.x - 1) / thread_block_size.x;
    timer.tick();
    KERNEL_WARN(critPoints<<<grid_size KERNEL_LAUNCH_SEPARATOR
                             thread_block_size>>>(dvv->computed,
                                                  dvv->index,
                                                  device_valences,
                                                  TV->nPoints,
                                                  max_VV_guess,
                                                  device_scalar_values,
                                                  device_CPC));
    timer.tick_announce();
    timer.label_next_interval("Retrieve results from GPU");
    timer.tick();
    CUDA_WARN(hipMemcpy(host_CPC, device_CPC, classes_size, hipMemcpyDeviceToHost));
    timer.tick();
    timer.label_next_interval("Export results from " CYAN_COLOR "Critical Points" RESET_COLOR " algorithm");
    timer.tick();
    export_classes(host_CPC, TV->nPoints, args);
    timer.tick_announce();
    if (host_CPC != nullptr) CUDA_WARN(hipHostFree(host_CPC));
    if (device_CPC != nullptr) CUDA_WARN(hipFree(device_CPC));
    if (device_valences != nullptr) CUDA_WARN(hipFree(device_valences));
    if (scalar_values != nullptr) CUDA_WARN(hipHostFree(scalar_values));
    if (device_scalar_values != nullptr) CUDA_WARN(hipFree(device_scalar_values));
    if (dvv != nullptr) free(dvv);
}

