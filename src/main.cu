#include "hip/hip_runtime.h"
// Other files in this repository
#include "argparse.h" // arguments type and parse()
#include "vtk_load.h" // TV_Data type and get_TV_from_VTK()
#include "cpu_extraction.h" // *_Data types and make_*() / elective_make_*()
#include "cuda_safety.h" // Cuda/Kernel safety wrappers
#include "cuda_extraction.h" // make_*_GPU()
#include "validate.h" // check_host_vs_device_*()
#include "metrics.h" // Timer class
#include "emoji.h" // Emoji definitions

__global__ void dummy_kernel(void) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
}

int main(int argc, char *argv[]) {
    Timer timer;
    arguments args;
    parse(argc, argv, args);
    timer.tick();
    timer.interval("Argument parsing");

    // GPU initialization
    {
        timer.label_next_interval("GPU context creation with dummy kernel");
        timer.tick();
        KERNEL_WARN(dummy_kernel<<<1 KERNEL_LAUNCH_SEPARATOR 1>>>());
        CUDA_ASSERT(hipDeviceSynchronize());
        timer.tick_announce();
        timer.label_next_interval("GPU trivial kernel launch");
        timer.tick();
        KERNEL_WARN(dummy_kernel<<<1 KERNEL_LAUNCH_SEPARATOR 1>>>());
        CUDA_ASSERT(hipDeviceSynchronize());
        timer.tick_announce();
    }

    // MANDATORY: TV (green) [from storage]
    std::cout << PUSHPIN_EMOJI << "Parsing vtu file: " << args.fileName
              << std::endl;
    timer.label_next_interval("TV from VTK");
    timer.tick();
    // Should utilize VTK API and then de-allocate all of its heap
    std::unique_ptr<TV_Data> TV = get_TV_from_VTK(args);
    timer.tick_announce();

    // MANDATORY: VE (red) [TV walk with semantic ordering to prevent dupes]
    // OPTIONAL: TE (green) [TV walk with semantic ordering to prevent dupes]
    std::cout << PUSHPIN_EMOJI << "Building edges..." << std::endl;
    std::unique_ptr<TE_Data> TE = std::make_unique<TE_Data>(TV->nCells);
    std::unique_ptr<VE_Data> VE = std::make_unique<VE_Data>(TV->nPoints);
    timer.tick();
    vtkIdType edgeCount;
    if (args.build_TE()) {
        timer.label_next_interval("TE and VE [CPU]");
        edgeCount = make_TE_and_VE(*TV, *TE, *VE);
    }
    else {
        timer.label_next_interval("VE [CPU]");
        edgeCount = make_VE(*TV, *VE);
    }
    timer.tick_announce();
    std::cout << OK_EMOJI << "Built " << edgeCount << " edges." << std::endl;

    // OPTIONAL: EV (green) [VE']
    if (args.build_EV()) {
        // CPU
        timer.label_next_interval("EV [CPU]");
        timer.tick();
        std::unique_ptr<EV_Data> EV = elective_make_EV(*VE, TV->nPoints,
                                                       edgeCount, args);
        timer.tick_announce();

        // GPU
        std::cout << PUSHPIN_EMOJI << "Using GPU to compute EV" << std::endl;
        timer.label_next_interval("EV [GPU]");
        timer.tick();
        std::unique_ptr<EV_Data> device_EV = make_EV_GPU(*VE, TV->nPoints,
                                                         edgeCount, args);
        timer.tick_announce();

        #ifdef VALIDATE_GPU
        // VALIDATION
        if (args.validate()) {
            timer.label_next_interval("Validate GPU EV");
            timer.tick();
            if (check_host_vs_device_EV(*EV, *device_EV)) {
                std::cout << OK_EMOJI << "GPU EV results validated by CPU"
                          << std::endl;
            }
            else {
                std::cerr << EXCLAIM_EMOJI
                          << "ALERT! GPU EV results do NOT match CPU results!"
                          << std::endl;
            }
            timer.tick_announce();
        }
        #endif
    }

    // OPTIONAL: TE (green) [TV x VE]
    if (args.build_TE()) {
        // CPU already prepared, GPU
        std::cout << PUSHPIN_EMOJI << "Using GPU to compute TE" << std::endl;
        std::cerr << EXCLAIM_EMOJI << "Not implemented yet" << std::endl;
        /*
        timer.label_next_interval("TE [GPU]");
        timer.tick();
        std::unique_ptr<TE_Data> device_TE = make_TE_GPU(*TE, *VE, n_edges, args);
        timer.tick_announce();

        #ifdef VALIDATE_GPU
        // VALIDATION
        if (args.validate()) {
            timer.label_next_interval("Validate GPU TE");
            timer.tick();
            if (check_host_vs_device_TE(*TE, *device_TE)) {
                std::cout << OK_EMOJI << "GPU TE results validated by CPU"
                          << std::endl;
            }
            else {
                std::cerr << EXCLAIM_EMOJI
                          << "ALERT! GPU TE results do NOT match CPU results!"
                          << std::endl;
            }
            timer.tick_announce();
        }
        #endif
        */

        // OPTIONAL: ET (red) [TE' == (TV x VE)']
        if (args.build_ET()) {
            // CPU
            timer.label_next_interval("ET [CPU]");
            timer.tick();
            // we can also get edgeStars from TE (ET)
            std::unique_ptr<ET_Data> ET = elective_make_ET(*TE, edgeCount, args);
            timer.tick_announce();

            // GPU
            std::cout << PUSHPIN_EMOJI << "Using GPU to compute ET" << std::endl;
            std::cerr << EXCLAIM_EMOJI << "Not implemented yet" << std::endl;
            /*
            timer.label_next_interval("ET [GPU]");
            timer.tick();
            std::unique_ptr<ET_Data> device_ET = make_ET_GPU(*TE, args);
            timer.tick_announce();

            #ifdef VALIDATE_GPU
            // VALIDATION
            if (args.validate()) {
                timer.label_next_interval("Validate GPU ET");
                timer.tick();
                if (check_host_vs_device_ET(*ET, *device_ET)) {
                    std::cout << OK_EMOJI << "GPU ET results validated by CPU"
                              << std::endl;
                }
                else {
                    std::cerr << EXCLAIM_EMOJI
                              << "ALERT! GPU ET results do NOT match CPU results!"
                              << std::endl;
                }
                timer.tick_announce();
            }
            #endif
            */
        }
    }

    // MANDATORY: VF (red) [TV walk with semantic ordering to prevent dupes]
    // OPTIONAL: TF (green) [TV walk with semantic ordering to prevent dupes]
    std::cout << PUSHPIN_EMOJI << "Building faces..." << std::endl;
    std::unique_ptr<TF_Data> TF = std::make_unique<TF_Data>(TV->nCells);
    std::unique_ptr<VF_Data> VF = std::make_unique<VF_Data>(TV->nPoints);
    vtkIdType faceCount;
    timer.tick();
    if (args.build_TF()) {
        timer.label_next_interval("TF and VF [CPU]");
        faceCount = make_TF_and_VF(*TV, *TF, *VF);
        /*
        for (vtkIdType i = 0; i < TV->nPoints; i++) {
            std::cout << "Vertex " << i << std::endl;
            for (auto face : (*VF)[i]) {
                std::cout << "\tFirst Face " << face.id << std::endl;
                break;
            }
            std::cout << "\tn_faces " << (*VF)[i].size() << std::endl;
        }
        */
    }
    else {
        timer.label_next_interval("VF [CPU]");
        faceCount = make_VF(*TV, *VF);
    }
    timer.tick_announce();
    std::cout << OK_EMOJI << "Built " << faceCount << " faces." << std::endl;

    // OPTIONAL: TF (green) [TV x VF]
    if (args.build_TF()) {
        std::cerr << EXCLAIM_EMOJI << "Not implemented yet" << std::endl;
        timer.label_next_interval("TF [GPU]");
        timer.tick();
        std::unique_ptr<TF_Data> device_TF = make_TF_GPU(*TV, *VF, TV->nPoints,
                                                         faceCount, TV->nCells,
                                                         args);
        timer.tick_announce();
        #ifdef VALIDATE_GPU
        if (args.validate()) {
            timer.label_next_interval("Validate GPU TF");
            timer.tick();
            if(check_host_vs_device_TF(*TF, *device_TF)) {
                std::cout << OK_EMOJI << "GPU TF results validated by CPU"
                          << std::endl;
            }
            else {
                std::cerr << EXCLAIM_EMOJI
                          << "ALERT! GPU TF results do NOT match CPU results!"
                          << std::endl;
            }
            timer.tick_announce();
        }
        #endif
    }

    // OPTIONAL: FV (green) [VF']
    if (args.build_FV()) {
        std::cerr << EXCLAIM_EMOJI << "Not implemented yet" << std::endl;
        /*
        timer.label_next_interval("FV [GPU]");
        timer.tick();
        std::unique_ptr<FV_Data> device_FV = make_FV_GPU(*VF, TV->nPoints,
                                                         faceCount, args);
        timer.tick_announce();
        #ifdef VALIDATE_GPU
        if (args.validate()) {
            timer.label_next_interval("Validate GPU FV");
            timer.tick();
            if(check_host_vs_device_FV(*FV, *device_FV)) {
                std::cout << OK_EMOJI << "GPU FV results validated by CPU"
                          << std::endl;
            }
            else {
                std::cerr << EXCLAIM_EMOJI
                          << "ALERT! GPU FV results do NOT match CPU results!"
                          << std::endl;
            }
            timer.tick_announce();
        }
        #endif
        */
    }

    // OPTIONAL: FE (green) [VF' x VE]
    if (args.build_FE()) {
        std::cerr << EXCLAIM_EMOJI << "Not implemented yet" << std::endl;
        /*
        timer.label_next_interval("FE [GPU]");
        timer.tick();
        std::unique_ptr<FE_Data> device_FE = make_FE_GPU(*VF, *VE, TV->nPoints,
                                                         faceCount, args);
        timer.tick_announce();
        #ifdef VALIDATE_GPU
        if (args.validate()) {
            timer.label_next_interval("Validate GPU FE");
            timer.tick();
            if(check_host_vs_device_FE(*FE, *device_FE)) {
                std::cout << OK_EMOJI << "GPU FE results validated by CPU"
                          << std::endl;
            }
            else {
                std::cerr << EXCLAIM_EMOJI
                          << "ALERT! GPU FE results do NOT match CPU results!"
                          << std::endl;
            }
            timer.tick_announce();
        }
        #endif
        */
    }

    // MIA: TT (yellow) [TV x TV']
    // MIA: FF (yellow) [TF' x TF]
    // MIA: EE (yellow) [EV' x VE]
    // MIA: VV (yellow) [TV' x TV]
    // MIA: FT (red) [(TV x VF)' | VF' x TV']
    // MIA: EF (red) [(TV x VE)' | VE' x TV']
    // MIA: VT (red) [TV']

    // Critical Points: FT = TF', VV = (V*') x (*V) for any of TV, FV, EV, VF, VE
    timer.tick(); // bonus tick -- open interval

    return 0;
}

